#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <string>
#include "../include/cuda_launch_config.hpp"
#include "../include/sirg.h"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <sys/time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define MAX_COLOR 1024
#define BLOCKSIZE 1024
#define ITERATIONS 1
#define FULL_MASK 0xffffffff
using namespace std;


#define CUDA_SAFE_CALL(ans) { cudaSafeCheck((ans), __FILE__, __LINE__);}
inline void cudaSafeCheck(hipError_t call, const char *file, int line, bool abort=true){
  if (call != hipSuccess){
    printf("Error: %s in file: %s at line: %d\n", hipGetErrorString(call), file, line);
    if (abort)
      exit(call);
  }
}

__global__ void firstFit(int* d_numVertices, const int* __restrict__ d_nonZeroIndices, const int* __restrict__ d_numColIndices, int* d_globalColoring, int* maxColor, long long int* adjColors, int* incSize){
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  int totalThreads = blockDim.x * gridDim.x;
  int VerticesPerThread = *d_numVertices/totalThreads;
  if (*d_numVertices%totalThreads != 0)
      VerticesPerThread++;
  int maxColorVal = (int)ceilf((*maxColor)/64.0);
  for (int id = threadID; VerticesPerThread > 0 && id < *d_numVertices; id += totalThreads, VerticesPerThread--){
      // get colors of neighbours
      int start = __ldg(d_numColIndices + id);
      int end = __ldg(d_numColIndices + id + 1);
      for (int i = start; i < end; i++){
        int color = d_globalColoring[__ldg(d_nonZeroIndices + i)];
        int adjColorIndex = blockIdx.x*(maxColorVal)*blockDim.x + threadIdx.x + blockDim.x*(color/64);
        if (color == *maxColor + 1)
          continue;
        adjColors[adjColorIndex] = adjColors[adjColorIndex] & ~(1LL << (color%64));
      }
      int i = 0;
      for (i = 0; i < maxColorVal; i++){
        int adjColorIndex = blockIdx.x*maxColorVal*blockDim.x + threadIdx.x + blockDim.x*i;
        int color = __ffsll(adjColors[adjColorIndex]);
        adjColors[adjColorIndex] = 0xffffffffffffffff;
        if (color){
          d_globalColoring[id] = i * 64 + color - 1;
          break;
        }
      }
      if (i == maxColorVal){
        *incSize = 1;
      }
      for (int j = i+1; j < maxColorVal; j++){
          int adjColorIndex = blockIdx.x*maxColorVal*blockDim.x + threadIdx.x + blockDim.x*j;
          adjColors[adjColorIndex] = 0xffffffffffffffff;
      }
  }
}

extern __shared__ int array[];

__global__ void conflictResolve(const int* __restrict__ d_nonZeroIndices,const int* __restrict__ d_numColIndices, int* d_globalColoring,const int* __restrict__ d_nextVertexIndices, int* inlist, int* inNumV, int* outlist, int* globalIndex, int* maxColor, long long int* adjColors, int* incSize){
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  int totalThreads = blockDim.x * gridDim.x;
  int VerticesPerThread = *inNumV/totalThreads;
  if (*inNumV%totalThreads != 0)
      VerticesPerThread++;
  int *threadBlockPrefixSum = array;
  int threadOutNumV = 0;
  int maxColorVal = (int)ceilf((*maxColor)/64.0);
  for (int vertex = threadID; VerticesPerThread > 0; vertex += totalThreads, VerticesPerThread--){
      // get colors of neighbours
      int recolorVertex;
      threadOutNumV = 0;
      if (vertex < *inNumV){
        int id = inlist[vertex];
        int nextVertex  = __ldg(d_nextVertexIndices + id);
        int end = __ldg(d_numColIndices + id + 1);
        int index;
        for (index = nextVertex; index < end; index++){
          if (d_globalColoring[id] == d_globalColoring[__ldg(d_nonZeroIndices + index)]){
            int start = __ldg(d_numColIndices + id);
            for (int i = start; i < end; i++){
              int color = d_globalColoring[__ldg(d_nonZeroIndices + i)];
              int adjColorIndex = blockIdx.x*maxColorVal*blockDim.x + threadIdx.x + blockDim.x*(color/64);
              adjColors[adjColorIndex] = adjColors[adjColorIndex] & ~(1LL << (color%64));
            }
            int i;
            for (i = 0; i < maxColorVal; i++){
              int adjColorIndex = blockIdx.x*maxColorVal*blockDim.x + threadIdx.x + blockDim.x*i;
              int color = __ffsll(adjColors[adjColorIndex]);
              adjColors[adjColorIndex] = 0xffffffffffffffff;
              if (color){
                d_globalColoring[id] = i * 64 + color - 1;
                recolorVertex = id;
                threadOutNumV = 1;
                break;
              }
            }
            if (i == maxColorVal){
              *incSize = 1;
              recolorVertex = id;
              threadOutNumV = 1;
            }
            for (int j = i+1; j < maxColorVal; j++){
                int adjColorIndex = blockIdx.x*maxColorVal*blockDim.x + threadIdx.x + blockDim.x*j;
                adjColors[adjColorIndex] = 0xffffffffffffffff;
            }
            break;
          }
        }
      }
      
      threadBlockPrefixSum[threadIdx.x] = threadOutNumV;
      int PrefixSumIndex = 0;
      int reduceOffset = 0;
      int lastThreadNum = (blockDim.x > 32) ? 31 : (blockDim.x-1);
      if (threadIdx.x%32 == lastThreadNum){
          thrust::exclusive_scan(thrust::device, threadBlockPrefixSum + threadIdx.x - lastThreadNum, threadBlockPrefixSum + threadIdx.x + 1, threadBlockPrefixSum + threadIdx.x - lastThreadNum);
          PrefixSumIndex = atomicAdd(globalIndex, threadBlockPrefixSum[threadIdx.x] + threadOutNumV);
      }
      __syncwarp(FULL_MASK);
      reduceOffset = threadBlockPrefixSum[threadIdx.x];
      threadBlockPrefixSum[threadIdx.x] = PrefixSumIndex;
      PrefixSumIndex = thrust::reduce(thrust::device, threadBlockPrefixSum + threadIdx.x - threadIdx.x%32, threadBlockPrefixSum + threadIdx.x - threadIdx.x%32 + 32, 0, thrust::maximum<int>());
      int arrayindex = PrefixSumIndex + reduceOffset;
      
      if (threadOutNumV == 1)
        outlist[arrayindex] = recolorVertex;
     
  }

}

__global__ void initializeColors(int* d_globalColoring, int* d_numVertices, int* maxDeg){
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  int totalThreads = blockDim.x * gridDim.x;
  int VerticesPerThread = *d_numVertices/totalThreads;
  if (*d_numVertices%totalThreads != 0)
      VerticesPerThread++;
  int color = (*maxDeg + 1);
  for (int id = threadID; VerticesPerThread > 0 && id < *d_numVertices; id += blockDim.x, VerticesPerThread--){
    d_globalColoring[id] = color;
  } 
}


__global__ void swapPointers(int* inNumV, int* globalIndex){
  *inNumV = *globalIndex;
  *globalIndex = 0;
}

int* colorGraphSIRG(int numVertices, int* nonZeroIndices, int* numColIndices, int numElems, int* nextVertexIndices, int maxDeg){

  double kernel_runtime[ITERATIONS];
  int numColors[ITERATIONS];
  int* globalColors;
  globalColors = (int *)malloc(sizeof(int)*numVertices);
  int maxColor;
  int blocks;
  for(int i = 0; i < ITERATIONS; i++){
    kernel_runtime[i] = 0;
    numColors[i] = 0;
    int *d_numVertices;
    int *d_nonZeroIndices, *d_numColIndices,*d_nextVertexIndices;
    int *d_globalColoring;
    hipEvent_t cudaStart, cudaEnd;
    hipEventCreate(&cudaStart);
    hipEventCreate(&cudaEnd);
    hipMalloc((void **)&d_numVertices, sizeof(int));
    hipMemcpy((void *)d_numVertices, (const void *) &numVertices, sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_nonZeroIndices, sizeof(int)*numElems);
    hipMemcpy((void *)d_nonZeroIndices, (const void *) nonZeroIndices, sizeof(int)*numElems, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_numColIndices, sizeof(int)*(numVertices + 1));
    hipMemcpy((void *)d_numColIndices, (const void *) numColIndices, sizeof(int)*(numVertices + 1), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_nextVertexIndices, sizeof(int)*numVertices);
    hipMemcpy((void *)d_nextVertexIndices, (const void *) nextVertexIndices, sizeof(int)*numVertices, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_globalColoring, sizeof(int)*numVertices);
    hipDeviceProp_t deviceProperty;
    hipGetDeviceProperties(&deviceProperty, 0);
    int numSMs = deviceProperty.multiProcessorCount;

    blocks = maximum_residency(conflictResolve, BLOCKSIZE, 0) * numSMs;
    blocks = numSMs;
    int* inlist, *outlist;
    hipMalloc((void **)&inlist, sizeof(int)*numVertices);
    thrust::sequence(thrust::device, inlist, inlist + numVertices);
    hipMalloc((void **)&outlist, sizeof(int)*numVertices);

    int *inNumV;
    hipMalloc((void **)&inNumV, sizeof(int));
    hipMemcpy((void *)inNumV, (const void *) &numVertices, sizeof(int), hipMemcpyHostToDevice);

    int *globalIndex;
    hipMalloc((void **)&globalIndex, sizeof(int));
    maxColor = min((int)pow(2, ceil(log(maxDeg + 1)/log(2))), 256);
    int* d_maxColor;
    hipMalloc((void **)&d_maxColor, sizeof(int));
    hipMemcpy((void *)d_maxColor, (const void *)&maxColor, sizeof(int), hipMemcpyHostToDevice);
    int *incSize;
    hipMalloc((void **)&incSize, sizeof(int));
    hipMemset((void *)incSize, 0, sizeof(int));
    long long int *d_adjColors;
    hipMalloc((void **)&d_adjColors, sizeof(long long int)*BLOCKSIZE*blocks*(ceil(maxColor/64.0)));
    hipMemset((void *)d_adjColors, 0xffffffff, sizeof(long long int)*BLOCKSIZE*blocks*(ceil(maxColor/64.0)));
    hipEventRecord(cudaStart, 0);
    initializeColors<<<blocks, BLOCKSIZE>>>(d_globalColoring, d_numVertices, d_maxColor);
    firstFit<<<blocks, BLOCKSIZE>>> (d_numVertices, d_nonZeroIndices, d_numColIndices, d_globalColoring, d_maxColor, d_adjColors, incSize);
    int hInNumV = numVertices;
    int hIncSize = 0;
    int iterNum = 0;
    while(hInNumV > 0){
      hipMemcpy((void *)&hIncSize, (const void *)incSize, sizeof(int), hipMemcpyDeviceToHost);
      if (hIncSize){
        hipFree(d_adjColors);
        maxColor *= 2;
        hipMemcpy((void *)d_maxColor, (const void *)&maxColor, sizeof(int), hipMemcpyHostToDevice);
        hipMalloc((void **)&(d_adjColors), sizeof(long long int)*BLOCKSIZE*blocks*(ceil(maxColor/64.0)));
        hipMemset((void *)d_adjColors, 0xffffffff, sizeof(long long int)*BLOCKSIZE*blocks*(ceil(maxColor/64.0)));
        hipMemset((void *)incSize, 0, sizeof(int));
      }

      if ((iterNum & 1) == 0){
        conflictResolve<<<blocks, BLOCKSIZE, BLOCKSIZE*4>>> (d_nonZeroIndices, d_numColIndices, d_globalColoring, d_nextVertexIndices, inlist, inNumV, outlist, globalIndex, d_maxColor, d_adjColors, incSize);
      }
      else{
        conflictResolve<<<blocks, BLOCKSIZE, BLOCKSIZE*4>>> (d_nonZeroIndices, d_numColIndices, d_globalColoring, d_nextVertexIndices, outlist, inNumV, inlist, globalIndex, d_maxColor, d_adjColors, incSize);
      }
      swapPointers<<<1,1>>>(inNumV, globalIndex);
      hipMemcpy((void *)&hInNumV, (const void *)inNumV, sizeof(int), hipMemcpyDeviceToHost);
      iterNum++;
    }
    hipDeviceSynchronize();
    hipEventRecord(cudaEnd, 0);
    hipFree(inlist);
    hipFree(outlist);
    hipFree(inNumV);
    hipFree(globalIndex);
    hipFree(d_maxColor);
    hipFree(incSize);
    hipFree(d_adjColors);
    hipMemcpy((void *)globalColors,(const void *) d_globalColoring, sizeof(int)*numVertices, hipMemcpyDeviceToHost);
    hipFree(d_globalColoring);
    hipFree(d_numVertices);
    hipFree(d_nonZeroIndices);
    hipFree(d_numColIndices);
    hipFree(d_nextVertexIndices);
    hipDeviceSynchronize();
    float time;
    hipEventElapsedTime(&time, cudaStart, cudaEnd);
    kernel_runtime[i] = time;
    numColors[i] = countColors(globalColors, numVertices);
    hipEventDestroy(cudaStart);
    hipEventDestroy(cudaEnd);
  }
    
  double averageKernelRuntime = 1.0;
  double averageColors = 1.0;

  for(int i = 0; i < ITERATIONS; i++){
    averageKernelRuntime *= kernel_runtime[i];
    averageColors *= numColors[i];
//    cout << "[ " << kernel_runtime[i] << " " << numColors[i] << " ] ";
  }
//  cout << endl;
  averageKernelRuntime = pow(averageKernelRuntime, 1.0/((double)ITERATIONS));
  averageColors = pow(averageColors, 1.0/((double)ITERATIONS));
  cout << fixed;
  cout << averageKernelRuntime << " " << averageColors << " " << sizeof(int)*BLOCKSIZE*blocks*(ceil(maxColor/64.0)) << endl;

//  cout << "Average Runtime: " << averageKernelRuntime << endl;
//  cout << "Average Colors: " << averageColors << endl;
//  cout << "Memory Used for adjColors array: " << sizeof(int)*BLOCKSIZE*blocks*(ceil(maxColor/64.0)) << " bytes" << endl;
  return globalColors;
}

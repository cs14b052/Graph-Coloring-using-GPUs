#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <math.h>
#include "../include/cuda_launch_config.hpp"
#include "../include/sirgl32.h"
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <sys/time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define MAX_COLOR 1024
#define BLOCKSIZE 1024
#define ITERATIONS 1
#define FULL_MASK 0xffffffff
using namespace std;


#define CUDA_SAFE_CALL(ans) { cudaSafeCheck((ans), __FILE__, __LINE__);}
inline void cudaSafeCheck(hipError_t call, const char *file, int line, bool abort=true){
  if (call != hipSuccess){
    printf("Error: %s in file: %s at line: %d\n", hipGetErrorString(call), file, line);
    if (abort)
      exit(call);
  }
}

__global__ void firstFit(int* d_numVertices, const int* __restrict__ d_nonZeroIndices, const int* __restrict__ d_numColIndices, int* d_globalColoring, int* maxColor, int* adjColors){
  
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  int totalThreads = blockDim.x * gridDim.x;
  int VerticesPerThread = *d_numVertices/totalThreads;
  if (*d_numVertices%totalThreads != 0)
      VerticesPerThread++;
  int maxColorVal = (int)ceilf((*maxColor)/32.0);
  for (int id = threadID; VerticesPerThread > 0 && id < *d_numVertices; id += totalThreads, VerticesPerThread--){
      // get colors of neighbours
      int start = __ldg(d_numColIndices + id);
      int end = __ldg(d_numColIndices + id + 1);
      for (int i = start; i < end; i++){
        int color = d_globalColoring[__ldg(d_nonZeroIndices + i)];
        int adjColorIndex = blockIdx.x*(maxColorVal)*blockDim.x + threadIdx.x + blockDim.x*(color/32);
        if (color == *maxColor + 1)
          continue;
        adjColors[adjColorIndex] = adjColors[adjColorIndex] & ~(1 << (color%32));
      }
      for (int i = 0; i < maxColorVal; i++){
        int adjColorIndex = blockIdx.x*maxColorVal*blockDim.x + threadIdx.x + blockDim.x*i;
        int color = __ffs(adjColors[adjColorIndex]);
        if (color){
          d_globalColoring[id] = i * 32 + color - 1;
          break;
        }
      }

      for (int j = 0; j < maxColorVal; j++){
          int adjColorIndex = blockIdx.x*maxColorVal*blockDim.x + threadIdx.x + blockDim.x*j;
          adjColors[adjColorIndex] = 0xffffffff;
      }
  }
}

extern __shared__ int array[];

__global__ void conflictResolve(const int* __restrict__ d_nonZeroIndices,const int* __restrict__ d_numColIndices, int* d_globalColoring,const int* __restrict__ d_nextVertexIndices, int* inlist, int* inNumV, int* outlist, int* globalIndex, int* maxColor, int* adjColors){
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  int totalThreads = blockDim.x * gridDim.x;
  int VerticesPerThread = *inNumV/totalThreads;
  if (*inNumV%totalThreads != 0)
      VerticesPerThread++;
  int *threadBlockPrefixSum = array;
  int threadOutNumV = 0;
  int maxColorVal = (int)ceilf((*maxColor)/32.0);
  for (int vertex = threadID; VerticesPerThread > 0; vertex += totalThreads, VerticesPerThread--){
      // get colors of neighbours
      int recolorVertex;
      threadOutNumV = 0;
      if (vertex < *inNumV){
        int id = inlist[vertex];
        int nextVertex  = __ldg(d_nextVertexIndices + id);
        int end = __ldg(d_numColIndices + id + 1);
        if (nextVertex == -1)
          nextVertex = end;
        int index;
        for (index = nextVertex; index < end; index++){
          if (d_globalColoring[id] == d_globalColoring[__ldg(d_nonZeroIndices + index)]){
            for (int j = 0; j < maxColorVal; j++){
                int adjColorIndex = blockIdx.x*maxColorVal*blockDim.x + threadIdx.x + blockDim.x*j;
                adjColors[adjColorIndex] = 0xffffffff;
            }

            int start = __ldg(d_numColIndices + id);
            for (int i = start; i < end; i++){
              int color = d_globalColoring[__ldg(d_nonZeroIndices + i)];
              int adjColorIndex = blockIdx.x*maxColorVal*blockDim.x + threadIdx.x + blockDim.x*(color/32);
              adjColors[adjColorIndex] = adjColors[adjColorIndex] & ~(1 << (color%32));
            }
            
            for (int i = 0; i < maxColorVal; i++){
              int adjColorIndex = blockIdx.x*maxColorVal*blockDim.x + threadIdx.x + blockDim.x*i;
              int color = __ffs(adjColors[adjColorIndex]);
              if (color){
                d_globalColoring[id] = i * 32 + color - 1;
                recolorVertex = id;
                threadOutNumV++;
                break;
              }
            }
            break;
          }
        } 
      }
      threadBlockPrefixSum[threadIdx.x] = threadOutNumV;
      int PrefixSumIndex = 0;
      int reduceOffset = 0;
      int lastThreadNum = (blockDim.x > 32) ? 31 : (blockDim.x-1);
      if (threadIdx.x%32 == lastThreadNum){
          thrust::exclusive_scan(thrust::device, threadBlockPrefixSum + threadIdx.x - lastThreadNum, threadBlockPrefixSum + threadIdx.x + 1, threadBlockPrefixSum + threadIdx.x - lastThreadNum);
          PrefixSumIndex = atomicAdd(globalIndex, threadBlockPrefixSum[threadIdx.x] + threadOutNumV);
      }
      __syncwarp(FULL_MASK);
      reduceOffset = threadBlockPrefixSum[threadIdx.x];
      threadBlockPrefixSum[threadIdx.x] = PrefixSumIndex;
      PrefixSumIndex = thrust::reduce(thrust::device, threadBlockPrefixSum + threadIdx.x - threadIdx.x%32, threadBlockPrefixSum + threadIdx.x - threadIdx.x%32 + 32, 0, thrust::maximum<int>());
      int arrayindex = PrefixSumIndex + reduceOffset;
      
      if (threadOutNumV == 1)
        outlist[arrayindex] = recolorVertex;
  }
}


__global__ void initializeColorsL32(int* d_globalColoring, int* d_numVertices, int* maxDeg){
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  int totalThreads = blockDim.x * gridDim.x;
  int VerticesPerThread = *d_numVertices/totalThreads;
  if (*d_numVertices%totalThreads != 0)
      VerticesPerThread++;
  for (int id = threadID; VerticesPerThread > 0 && id < *d_numVertices; id += blockDim.x, VerticesPerThread--){
    int color = (*maxDeg + 1);
    d_globalColoring[id] = color;
  } 
}

__global__ void swapPointersL32(int* inNumV, int* globalIndex){
  *inNumV = *globalIndex;
  *globalIndex = 0;
}


int* colorGraphSIRGL32(int numVertices, int* nonZeroIndices, int* numColIndices, int numElems, int* nextVertexIndices, int maxDeg){

  double kernel_runtime[ITERATIONS];
  int numColors[ITERATIONS];
  int* globalColors;
  globalColors = (int *)malloc(sizeof(int)*numVertices);
  int maxColor;
  int blocks;
  for(int i = 0; i < ITERATIONS; i++){
    kernel_runtime[i] = 0;
    numColors[i] = 0;
    int *d_numVertices;
    int *d_nonZeroIndices, *d_numColIndices,*d_nextVertexIndices;
    int *d_globalColoring;
    hipEvent_t cudaStart, cudaEnd;
    hipEventCreate(&cudaStart);
    hipEventCreate(&cudaEnd);
    hipMalloc((void **)&d_numVertices, sizeof(int));
    hipMemcpy((void *)d_numVertices, (const void *) &numVertices, sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_nonZeroIndices, sizeof(int)*numElems);
    hipMemcpy((void *)d_nonZeroIndices, (const void *) nonZeroIndices, sizeof(int)*numElems, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_numColIndices, sizeof(int)*(numVertices + 1));
    hipMemcpy((void *)d_numColIndices, (const void *) numColIndices, sizeof(int)*(numVertices + 1), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_nextVertexIndices, sizeof(int)*numVertices);
    hipMemcpy((void *)d_nextVertexIndices, (const void *) nextVertexIndices, sizeof(int)*numVertices, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_globalColoring, sizeof(int)*numVertices);

    hipDeviceProp_t deviceProperty;
    hipGetDeviceProperties(&deviceProperty, 0);
    int numSMs = deviceProperty.multiProcessorCount;

    blocks = maximum_residency(conflictResolve, BLOCKSIZE, 0) * numSMs;
    blocks = numSMs;
    int* inlist, *outlist;
    hipMalloc((void **)&inlist, sizeof(int)*numVertices);
    thrust::sequence(thrust::device, inlist, inlist + numVertices);
    hipMalloc((void **)&outlist, sizeof(int)*numVertices);

    int *inNumV;
    hipMalloc((void **)&inNumV, sizeof(int));
    hipMemcpy((void *)inNumV, (const void *) &numVertices, sizeof(int), hipMemcpyHostToDevice);

    int *globalIndex;
    hipMalloc((void **)&globalIndex, sizeof(int));


    maxColor = pow(2, ceil(log(maxDeg + 1)/log(2)));
    int* d_maxColor;
    hipMalloc((void **)&d_maxColor, sizeof(int));
    hipMemcpy((void *)d_maxColor, (const void *)&maxColor, sizeof(int), hipMemcpyHostToDevice);
    
    int *d_adjColors;
    hipMalloc((void **)&d_adjColors, sizeof(int)*BLOCKSIZE*blocks*(ceil(maxColor/32.0)));
    hipMemset((void *)d_adjColors, 0xffffffff, sizeof(int)*BLOCKSIZE*blocks*(ceil(maxColor/32.0)));
    hipEventRecord(cudaStart, 0);

    initializeColorsL32<<<blocks, BLOCKSIZE>>>(d_globalColoring, d_numVertices, d_maxColor);
    firstFit<<<blocks, BLOCKSIZE>>> (d_numVertices, d_nonZeroIndices, d_numColIndices, d_globalColoring, d_maxColor, d_adjColors);
    int hInNumV = numVertices;
    int iterNum = 0;
    while(hInNumV > 0){
      if ((iterNum & 1) == 0){
        conflictResolve<<<blocks, BLOCKSIZE, BLOCKSIZE*4>>> (d_nonZeroIndices, d_numColIndices, d_globalColoring, d_nextVertexIndices, inlist, inNumV, outlist, globalIndex, d_maxColor, d_adjColors);
      }
      else{
        conflictResolve<<<blocks, BLOCKSIZE, BLOCKSIZE*4>>> (d_nonZeroIndices, d_numColIndices, d_globalColoring, d_nextVertexIndices, outlist, inNumV, inlist, globalIndex, d_maxColor, d_adjColors);
      }
       swapPointersL32<<<1,1>>>(inNumV, globalIndex);
       hipMemcpy((void *)&hInNumV, (const void *)inNumV, sizeof(int), hipMemcpyDeviceToHost);
      iterNum++;
    }
    hipDeviceSynchronize();
    hipEventRecord(cudaEnd, 0);
    hipFree(inlist);
    hipFree(outlist);
    hipFree(inNumV);
    hipFree(globalIndex);
    hipFree(d_maxColor);
    hipFree(d_adjColors);
    hipMemcpy((void *)globalColors,(const void *) d_globalColoring, sizeof(int)*numVertices, hipMemcpyDeviceToHost);
    hipFree((void *)d_globalColoring);
    hipFree(d_numVertices);
    hipFree(d_nonZeroIndices);
    hipFree(d_numColIndices);
    hipFree(d_nextVertexIndices);
    hipDeviceSynchronize();
    float time;
    hipEventElapsedTime(&time, cudaStart, cudaEnd);
    kernel_runtime[i] = time;
    numColors[i] = countColors(globalColors, numVertices);
    hipEventDestroy(cudaStart);
    hipEventDestroy(cudaEnd);
  }
    
  double averageKernelRuntime = 1.0;
  double averageColors = 1.0;

  for(int i = 0; i < ITERATIONS; i++){
    averageKernelRuntime *= kernel_runtime[i];
    averageColors *= numColors[i];
//    cout << "[ " << kernel_runtime[i] << " " << numColors[i] << " ] ";
  }
//  cout << endl;
  averageKernelRuntime = pow(averageKernelRuntime, 1.0/((double)ITERATIONS));
  averageColors = pow(averageColors, 1.0/((double)ITERATIONS));
  cout << fixed;
  cout << averageKernelRuntime << " " << averageColors << " " << sizeof(int)*BLOCKSIZE*blocks*(ceil(maxColor/32.0)) << endl;
//  cout << "Average Runtime: " << averageKernelRuntime << endl;
//  cout << "Average Colors: " << averageColors << endl;
//  cout << "Memory Used for adjColors array: " << sizeof(int)*BLOCKSIZE*blocks*(ceil(maxColor/32.0)) << " bytes" << endl;
  return globalColors;
}

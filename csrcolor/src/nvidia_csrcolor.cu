#include <stdio.h>
#include <iostream>
#include <math.h>
#include "../include/nvidia_csrcolor.h"
#include <sys/time.h>
#include <hipsparse.h>
#define BLOCKSIZE 1024
using namespace std;

int* colorGraph(int numVertices, int* nonZeroIndices, int* numColIndices, int numElems, int* nextVertexIndices, int maxDeg){

  int* globalColors;
  globalColors = (int *)malloc(sizeof(int)*numVertices);

  int *d_nonZeroIndices, *d_numColIndices;
  int *d_globalColoring;
  float* d_csrVal;
  int numColors = 0;
  float fractionToColor = 1.0;
  hipEvent_t cudaStart, cudaEnd;
  hipEventCreate(&cudaStart);
  hipEventCreate(&cudaEnd);
  hipMalloc((void **)&d_nonZeroIndices, sizeof(int)*numElems);
  hipMemcpy((void *)d_nonZeroIndices, (const void *) nonZeroIndices, sizeof(int)*numElems, hipMemcpyHostToDevice);
  hipMalloc((void **)&d_numColIndices, sizeof(int)*(numVertices + 1));
  hipMemcpy((void *)d_numColIndices, (const void *) numColIndices, sizeof(int)*(numVertices + 1), hipMemcpyHostToDevice);
  hipMalloc((void **)&d_globalColoring, sizeof(int)*numVertices);
  hipMemset((void *)d_globalColoring, -1, sizeof(int)*numVertices);
  hipMalloc((void **)&d_csrVal, sizeof(float)*numElems);
  
  hipsparseStatus_t status;

  hipsparseHandle_t handle;
  status = hipsparseCreate(&handle);
  hipsparseMatDescr_t descriptor;
  status = hipsparseCreateMatDescr(&descriptor);
  hipsparseColorInfo_t info;
  status = hipsparseCreateColorInfo(&info);
  
  hipEventRecord(cudaStart, 0);
  status = hipsparseScsrcolor(handle, numVertices, numElems, descriptor, d_csrVal, d_numColIndices, d_nonZeroIndices, &fractionToColor, &numColors, d_globalColoring, NULL, info);
  hipDeviceSynchronize();
  hipEventRecord(cudaEnd, 0);

  hipMemcpy((void *)globalColors,(const void *) d_globalColoring, sizeof(int)*numVertices, hipMemcpyDeviceToHost);
  
  hipFree((void *)d_globalColoring);
  hipFree(d_nonZeroIndices);
  hipFree(d_numColIndices);
  hipFree(d_csrVal);
  hipDeviceSynchronize();
  
  float time;
  hipEventElapsedTime(&time, cudaStart, cudaEnd);
  /*
  cout << "Cuda Kernel Runtime: " << time << endl;
  cout << "Number of Colors: " << numColors << endl;
  */
  cout << time << " " << numColors << endl;
  return globalColors;

} 
